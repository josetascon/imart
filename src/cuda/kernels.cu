#include "hip/hip_runtime.h"
/*
* @Author: jose
* @Date:   2020-08-24 00:00:00
* @Last Modified by:   jose
* @Last Modified time: 2020-08-24 00:00:00
*/

// local libs
#include "kernels.cuh"

// ===========================================
// Check Errors
// ===========================================
#define imart_assert_kernel(status, msg) \
    imart_assert_kernel_error((status), __FILE__, __LINE__, msg);

void imart_assert_kernel_error(hipError_t code, const char *file, int line, const char* msg, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\n******* CUDA Error *******"\
                    "\n[Error] Information:\t%s"\
                    "\n[Error] Error code:\t%i"\
                    "\n[Error] Description:\t%s"\
                    "\n[Error] File:\t\t%s"\
                    "\n[Error] Line:\t\t%d\n",
                    msg, code, hipGetErrorString(code), file, line);
      if (abort) exit(code); 
   };
};

// ===========================================
// Kernels
// ===========================================


// ===========================================
// Data Kernels
// ===========================================
template <typename type>
__global__ void kernel_assign(type * vin, type value, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vin[i] = value;
};

template <typename type>
__global__ void kernel_copy(const type * vin, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin[i];
};

template <typename typein, typename typeout>
__global__ void kernel_cast(const typein * vin, typeout * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = typeout(vin[i]);
};


// ===========================================
// Vector Kernels
// ===========================================
template <typename type>
__global__ void kernel_add_scalar(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin[i] + scalar;
};

template <typename type>
__global__ void kernel_sub_scalar(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin[i] - scalar;
};

template <typename type>
__global__ void kernel_sub_scalar_inv(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = scalar - vin[i];
};

template <typename type>
__global__ void kernel_mul_scalar(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin[i] * scalar;
};

template <typename type>
__global__ void kernel_div_scalar(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin[i] / scalar;
};

template <typename type>
__global__ void kernel_div_scalar_inv(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = scalar / vin[i];
};

template <typename type>
__global__ void kernel_pow_scalar(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = pow( vin[i], scalar );
};

template <typename type>
__global__ void kernel_pow_scalar_inv(const type * vin, type * vout, type scalar, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = pow( scalar, vin[i] );
};

template <typename type>
__global__ void kernel_add(const type * vin1, const type * vin2, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin1[i] + vin2[i];
};

template <typename type>
__global__ void kernel_sub(const type * vin1, const type * vin2, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin1[i] - vin2[i];
};

template <typename type>
__global__ void kernel_mul(const type * vin1, const type * vin2, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin1[i] * vin2[i];
};

template <typename type>
__global__ void kernel_div(const type * vin1, const type * vin2, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = vin1[i] / vin2[i];
};

template <typename type>
__global__ void kernel_pow(const type * vin1, const type * vin2, type * vout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) vout[i] = pow( vin1[i], vin2[i] );
};

// ===========================================
// Reduction Kernels
// ===========================================
template <typename type>
__global__ void kernel_sum(const type *vin, type *vout, int n)
{
    __shared__ type sdata[256]; // Warning, threads should be 256
    unsigned int iii = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    type sum = 0;

    for (unsigned int i = iii; i < n; i += gridDim.x * blockDim.x)
    {
        sum += vin[i];
    };
    
    sdata[tid] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        };
        __syncthreads();
    };

    if (tid == 0) vout[blockIdx.x] = sdata[0];
};


template <typename type>
__global__ void kernel_min(const type *vin, type *vout, int n)
{
    __shared__ type sdata[256];
    unsigned int iii = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    type thread_result = vin[0];

    for (unsigned int i = iii; i < n; i += gridDim.x * blockDim.x)
    {
        type tmp = vin[i];
        thread_result = thread_result < tmp ? thread_result : tmp;
    };

    sdata[tid] = thread_result;
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] =  sdata[tid] < sdata[tid + s]? sdata[tid] : sdata[tid + s];
        };
        __syncthreads();
    };

    if (tid == 0) vout[blockIdx.x] = sdata[0];
};

template <typename type>
__global__ void kernel_max(const type *vin, type *vout, int n)
{
    __shared__ type sdata[256];
    unsigned int iii = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    type thread_result = vin[0];

    for (unsigned int i = iii; i < n; i += gridDim.x * blockDim.x)
    {
        type tmp = vin[i];
        thread_result = thread_result > tmp ? thread_result : tmp;
    };

    sdata[tid] = thread_result;
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] =  sdata[tid] > sdata[tid + s]? sdata[tid] : sdata[tid + s];
        };
        __syncthreads();
    };

    if (tid == 0) vout[blockIdx.x] = sdata[0];
};

// ===========================================
// Image Kernels
// ===========================================
template <typename type>
__global__ void kernel_pad_2d(const type * vin, type * vout, int start0, int start1,
                              int end0, int end1, int n0, int n1)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int wo = n0+start0+end0;

    if (i < n0 && j < n1) // width = n0, heigth = n1
    {
        vout[start0+i + (start1+j)*wo] = vin[i + j*n0];
    };
};

template <typename type>
__global__ void kernel_unpad_2d(const type * vin, type * vout, int start0, int start1,
                              int end0, int end1, int n0, int n1)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int wo = n0+start0+end0;

    if (i < n0 && j < n1) // width = n0, heigth = n1
    {
        vout[i + j*n0] = vin[start0+i + (start1+j)*wo];
    };
};

template <typename type>
__global__ void kernel_pad_3d(const type * vin, type * vout, int start0, int start1, int start2,
                              int end0, int end1, int end2, int n0, int n1, int n2)
{
    // int blockIdx_z = __float2int_rd(blockIdx.y * invBlocksInY);
    // int blockIdx_y = blockIdx.y - (blockIdx_z * blocksInY);
    // int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    // int j = (blockIdx_y * blockDim.y) + threadIdx.y;
    // int k = (blockIdx_z * blockDim.z) + threadIdx.z;
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    int wo = n0+start0+end0; //vout size
    int ho = n1+start1+end1; //vout size

    if (i < n0 && j < n1 && k < n2) // width = n0, height = n1, depth = n2
    {
        vout[start0+i + (start1+j)*wo + (start2+k)*wo*ho] = vin[i + j*n0 + k*n0*n1];
    };
};

template <typename type>
__global__ void kernel_unpad_3d(const type * vin, type * vout, int start0, int start1, int start2,
                              int end0, int end1, int end2, int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    int wo = n0+start0+end0; //vout size
    int ho = n1+start1+end1; //vout size

    if (i < n0 && j < n1 && k < n2) // width = n0, height = n1, depth = n2
    {
        vout[i + j*n0 + k*n0*n1] = vin[start0+i + (start1+j)*wo + (start2+k)*wo*ho];
    };
};

template <typename type>
__global__ void kernel_grid_2d( type * x, type * y, double * sod, 
                                int n0, int n1)
{
    // consider sod conversion to float to support all gpu
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    double c0 = sod[0]; double c1 = sod[1];
    double o0 = sod[2]; double o1 = sod[3];
    double d0 = sod[4]; double d1 = sod[5];
    double d2 = sod[6]; double d3 = sod[7];

    if (i < n0 && j < n1) // width = n0, heigth = n1
    {
        x[i+j*n0] = (type)(d0*c0*i + d1*c1*j + o0);
        y[i+j*n0] = (type)(d2*c0*i + d3*c1*j + o1);
    };
};

template <typename type>
__global__ void kernel_grid_3d( type * x, type * y, type * z, double * sod, 
                                int n0, int n1, int n2)
{
    // consider sod conversion to float to support all gpu
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    double c0 = sod[0]; double c1 = sod[1]; double c2 = sod[2];
    double o0 = sod[3]; double o1 = sod[4]; double o2 = sod[5];
    double d0 = sod[6]; double d1 = sod[7]; double d2 = sod[8];
    double d3 = sod[9]; double d4 = sod[10]; double d5 = sod[11];
    double d6 = sod[12]; double d7 = sod[13]; double d8 = sod[14];

    if (i < n0 && j < n1 && k < n2) // width = n0, height = n1, depth = n2
    {
        x[i + j*n0 + k*n0*n1] = (type)(d0*c0*i + d1*c1*j + d2*c2*k + o0);
        y[i + j*n0 + k*n0*n1] = (type)(d3*c0*i + d4*c1*j + d5*c2*k + o1);
        z[i + j*n0 + k*n0*n1] = (type)(d6*c0*i + d7*c1*j + d8*c2*k + o2);
    };
};

template <typename type>
__global__ void kernel_affine_2d( const type * xin, const type * yin, 
                                  type * xout, type * yout, 
                                  const type * param, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xy equal size
    type a0 = param[0]; type a1 = param[1];
    type a2 = param[2]; type a3 = param[3];
    type t0 = param[4]; type t1 = param[5];
    if (i < n)
    {
        xout[i] = (type)(a0*xin[i] + a1*yin[i] + t0);
        yout[i] = (type)(a2*xin[i] + a3*yin[i] + t1);
    };
};

template <typename type>
__global__ void kernel_affine_3d( const type * xin, const type * yin, const type * zin,
                                  type * xout, type * yout, type * zout,
                                  const type * param, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xyz equal size
    type a0 = param[0]; type a1 = param[1]; type a2 = param[2];
    type a3 = param[3]; type a4 = param[4]; type a5 = param[5];
    type a6 = param[6]; type a7 = param[7]; type a8 = param[8];
    type t0 = param[9]; type t1 = param[10]; type t2 = param[11];
    if (i < n)
    {
        xout[i] = (type)(a0*xin[i] + a1*yin[i] + a2*zin[i] + t0);
        yout[i] = (type)(a3*xin[i] + a4*yin[i] + a5*zin[i] + t1);
        zout[i] = (type)(a6*xin[i] + a7*yin[i] + a8*zin[i] + t2);
    };
};

template <typename type>
__global__ void kernel_affine_sod_2d( const type * xin, const type * yin,
                                      type * xout, type * yout,
                                      const double * sod, int n)
{
    // consider sod conversion to float to support all gpu
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xy equal size
    double c0 = sod[0]; double c1 = sod[1];
    double o0 = sod[2]; double o1 = sod[3];
    double d0 = sod[4]; double d1 = sod[5];
    double d2 = sod[6]; double d3 = sod[7];
    if (i < n)
    {
        xout[i] = (type)(d0*c0*xin[i] + d1*c1*yin[i] + o0);
        yout[i] = (type)(d2*c0*xin[i] + d3*c1*yin[i] + o1);
    }
};

template <typename type>
__global__ void kernel_affine_sod_3d( const type * xin, const type * yin, const type * zin,
                                      type * xout, type * yout, type * zout,
                                      const double * sod, int n)
{
    // consider sod conversion to float to support all gpu
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xyz equal size
    double c0 = sod[0]; double c1 = sod[1]; double c2 = sod[2];
    double o0 = sod[3]; double o1 = sod[4]; double o2 = sod[5];
    double d0 = sod[6]; double d1 = sod[7]; double d2 = sod[8];
    double d3 = sod[9]; double d4 = sod[10]; double d5 = sod[11];
    double d6 = sod[12]; double d7 = sod[13]; double d8 = sod[14];
    if (i < n)
    {
        xout[i] = (type)(d0*c0*xin[i] + d1*c1*yin[i] + d2*c2*zin[i] + o0);
        yout[i] = (type)(d3*c0*xin[i] + d4*c1*yin[i] + d5*c2*zin[i] + o1);
        zout[i] = (type)(d6*c0*xin[i] + d7*c1*yin[i] + d8*c2*zin[i] + o2);
    };
};

template <typename type>
__global__ void kernel_dfield_2d( const type * xin, const type * yin,   // grid coordinates
                                  const type * x, const type * y,       // vector field
                                  type * xout, type * yout, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xy equal size
    if (i < n)
    {
        xout[i] = xin[i] + x[i];
        yout[i] = yin[i] + y[i];
    };
};

template <typename type>
__global__ void kernel_dfield_3d( const type * xin, const type * yin, const type * zin, // grid coordinates
                                  const type * x, const type * y, const type * z,       // vector field
                                  type * xout, type * yout, type * zout,                // output coordinates
                                  int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // one dimension, buffer in and out xy equal size
    if (i < n)
    {
        xout[i] = xin[i] + x[i];
        yout[i] = yin[i] + y[i];
        zout[i] = zin[i] + z[i];
    };
};

template <typename type>
__global__ void kernel_nearest_interpolation_2d( const type * xo, const type * yo,
                                                 const type * imgr, type * imgo,
                                                 int w, int h,   //img ref width and height
                                                 int n0, int n1) //img out dims
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < n0 && j < n1)
    {
        int x = round(xo[i + j*n0]);
        int y = round(yo[i + j*n0]);
        if(x >= 0 && x < w && y >= 0 && y < h)
        {
            imgo[i + j*n0] = imgr[x + y*w];
        };
    };
};

template <typename type>
__global__ void kernel_nearest_interpolation_3d( const type * xo, const type * yo, const type * zo, 
                                                 const type * imgr, type * imgo,
                                                int w, int h, int l,    //img ref width, height and length
                                                int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    if (i < n0 && j < n1 && k < n2)
    {
        int x = round(xo[i + j*n0 + k*n0*n1]);
        int y = round(yo[i + j*n0 + k*n0*n1]);
        int z = round(yo[i + j*n0 + k*n0*n1]);
        if(x >= 0 && x < w && y >= 0 && y < h && z >= 0 && z < l)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[x + y*w + z*w*h];
        };
    };
};

template <typename type>
__global__ void kernel_linear_interpolation_2d( const type * xo, const type * yo,
                                                const type * imgr, type * imgo,
                                                int w, int h,   //img ref width and height
                                                int n0, int n1) //img out dims
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < n0 && j < n1)
    {
        type xt = xo[i + j*n0];
        type yt = yo[i + j*n0];
        int x = floor(xt);
        int y = floor(yt);
        if(x >= 0 && x < w && y >= 0 && y < h - 1)
        {
            // __shared__ iv[4];
            type iv[4] = {imgr[x+y*w], imgr[x+1+y*w], imgr[x+(y+1)*w], imgr[x+1+(y+1)*w]};
            type dx = xt - (type)x;
            type dy = yt - (type)y;
            type dxdy = dx*dy;
            type r = iv[0]*(1-dx-dy+dxdy) + iv[1]*(dx-dxdy) + iv[2]*(dy-dxdy) + iv[3]*dxdy;
            imgo[i + j*n0] = r;
        }
        else if(x >= 0 && x < w && y == h - 1) // border case
        {
            type iv[2] = {imgr[x+y*w], imgr[x+1+y*w]};
            type dx = xt - (type)x;
            type r = iv[0]*(1-dx) + iv[1]*(dx);
            imgo[i + j*n0] = r;
        };
    };
};

template <typename type>
__global__ void kernel_linear_interpolation_3d( const type * xo, const type * yo, const type * zo,
                                                const type * imgr, type * imgo,
                                                int w, int h, int l, //img ref width, height and length
                                                int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    if (i < n0 && j < n1 && k < n2)
    {
        type xt = xo[i + j*n0 + k*n0*n1];
        type yt = yo[i + j*n0 + k*n0*n1];
        type zt = zo[i + j*n0 + k*n0*n1];
        int x = floor(xt);
        int y = floor(yt);
        int z = floor(zt);
        if(x >= 0 && x < w && y >= 0 && y < h && z >= 0 && z < l-1)
        {
            type iv[4] = {imgr[x+y*w+z*w*h], imgr[x+1+y*w+z*w*h], imgr[x+(y+1)*w+z*w*h], imgr[x+1+(y+1)*w+z*w*h]};
            type iw[4] = {imgr[x+y*w+(z+1)*w*h], imgr[x+1+y*w+(z+1)*w*h], imgr[x+(y+1)*w+(z+1)*w*h], imgr[x+1+(y+1)*w+(z+1)*w*h]};
            type dx = xt - (type)x;
            type dy = yt - (type)y;
            type dxdy = dx*dy;
            type rv = iv[0]*(1-dx-dy+dxdy) + iv[1]*(dx-dxdy) + iv[2]*(dy-dxdy) + iv[3]*dxdy;
            type rw = iw[0]*(1-dx-dy+dxdy) + iw[1]*(dx-dxdy) + iw[2]*(dy-dxdy) + iw[3]*dxdy;
            type dz = zt - (type)z;
            type r = rv*(1-dz) + rw*dz;
            imgo[i + j*n0 + k*n0*n1] = r;
        }
        else if(x >= 0 && x < w && y >= 0 && y < h && z == l-1) // border case
        {
            type iv[4] = {imgr[x+y*w+z*w*h], imgr[x+1+y*w+z*w*h], imgr[x+(y+1)*w+z*w*h], imgr[x+1+(y+1)*w+z*w*h]};
            type dx = xt - (type)x;
            type dy = yt - (type)y;
            type dxdy = dx*dy;
            type rv = iv[0]*(1-dx-dy+dxdy) + iv[1]*(dx-dxdy) + iv[2]*(dy-dxdy) + iv[3]*dxdy;
            imgo[i + j*n0 + k*n0*n1] = rv;
        };
    };
};

template <typename type>
__global__ void kernel_gradientx( const type * imgr, type * imgo, 
                                  int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    if (i < n0 && j < n1 && (k == 0 || k < n2))
    {
        if(i == 0)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i+1 + j*n0 + k*n0*n1] - imgr[i + j*n0 + k*n0*n1];
        }
        else if(i == n0 - 1)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i + j*n0 + k*n0*n1] - imgr[i-1 + j*n0 + k*n0*n1];
        }
        else
        {
            imgo[i + j*n0 + k*n0*n1] = 0.5*imgr[i+1 + j*n0 + k*n0*n1] - 0.5*imgr[i-1 + j*n0 + k*n0*n1];
        };
    };
};

template <typename type>
__global__ void kernel_gradienty( const type * imgr, type * imgo, 
                                  int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    if (i < n0 && j < n1 && (k == 0 || k < n2))
    {
        if(j == 0)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i + (j+1)*n0 + k*n0*n1] - imgr[i + j*n0 + k*n0*n1];
        }
        else if(j == n1 - 1)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i + j*n0 + k*n0*n1] - imgr[i + (j-1)*n0 + k*n0*n1];
        }
        else
        {
            imgo[i + j*n0 + k*n0*n1] = 0.5*imgr[i + (j+1)*n0 + k*n0*n1] - 0.5*imgr[i + (j-1)*n0 + k*n0*n1];
        };
    };
};

template <typename type>
__global__ void kernel_gradientz( const type * imgr, type * imgo, 
                                  int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;

    if (i < n0 && j < n1 && k < n2)
    {
        if(k == 0)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i + j*n0 + (k+1)*n0*n1] - imgr[i + j*n0 + k*n0*n1];
        }
        else if(k == n2 - 1)
        {
            imgo[i + j*n0 + k*n0*n1] = imgr[i + j*n0 + k*n0*n1] - imgr[i + j*n0 + (k-1)*n0*n1];
        }
        else
        {
            imgo[i + j*n0 + k*n0*n1] = 0.5*imgr[i + j*n0 + (k+1)*n0*n1] - 0.5*imgr[i + j*n0 + (k-1)*n0*n1];
        };
    };
};

template <typename type>
__global__ void kernel_convolution_2d( const type * imgr, const type * kern, //kernel width
                                       type * imgo, int kwidth, int n0, int n1)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int off = floor(kwidth/2.0);

    if(i >= off && i < n0 - off && j >= off && j < n1 - off)
    {
        type sum = 0;
        for (int p = 0; p < kwidth; p++)
        {
            for (int q = 0; q < kwidth; q++)
            {
                sum += imgr[i+p-off + (j+q-off)*n0] * kern[p*kwidth + q];
            };
        };
        imgo[i + j*n0] = sum;
    };
};

template <typename type>
__global__ void kernel_convolution_3d( const type * imgr, const type * kern, //kernel width
                                       type * imgo, int kwidth, int n0, int n1, int n2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int k = (blockIdx.z * blockDim.z) + threadIdx.z;
    
    int off = floor(kwidth/2.0);

    if(i >= off && i < n0 - off && j >= off && j < n1 - off && k >= off && k < n2 - off)
    {
        type sum = 0;
        for (int r = 0; r < kwidth; r++)
        {
            for (int p = 0; p < kwidth; p++)
            {
                for (int q = 0; q < kwidth; q++)
                {
                    sum += imgr[i+p-off + (j+q-off)*n0 + (k-off)*n0*n1] * kern[r*kwidth*kwidth + p*kwidth + q];
                };
            };
        };
        imgo[i + j*n0 + k*n0*n1] = sum;
    };
};


















// ===========================================
// Kernels Calls
// ===========================================


// ===========================================
// Data Kernels
// ===========================================
template <typename type>
void cuda_kernel_assign( std::vector<int> & grid, std::vector<int> & block, 
                         type * vin, type value, int n )
{
    // printf("kernel assign init\n");
    // printf("block: [%i, %i, %i]\n", block[0], block[1] , block[2]);
    // printf("grid: [%i, %i, %i]\n", grid[0], grid[1] , grid[2]);
    // printf("address: %x\n", vin);
    // printf("value: %f\n", value);
    // printf("size: %i\n", n);

    dim3 grd(grid[0]);
    dim3 blk(block[0]);

    kernel_assign<<<grd,blk>>>(vin, value, n);
    // kernel_assign<type><<<grd,blk>>>(vin, value, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel assign" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel assign" );
    // printf("kernel assign finish\n");
};

template <typename type>
void cuda_kernel_copy( std::vector<int> & grid, std::vector<int> & block,
                       const type * vin, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_copy<<<grd,blk>>>(vin, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel copy" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel copy" );
};

template <typename typein, typename typeout>
void cuda_kernel_cast( std::vector<int> & grid, std::vector<int> & block, 
                       const typein * vin, typeout * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_cast<typein,typeout><<<grd,blk>>>(vin, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel cast" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel cast" );
};

// ===========================================
// Vector Kernels
// ===========================================
template <typename type>
void cuda_kernel_add_scalar( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_add_scalar<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel add scalar" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel add scalar" );
};

template <typename type>
void cuda_kernel_sub_scalar( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_sub_scalar<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel sub scalar" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel sub scalar" );
};

template <typename type>
void cuda_kernel_sub_scalar_inv( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_sub_scalar_inv<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel sub scalar inv" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel sub scalar inv" );
};

template <typename type>
void cuda_kernel_mul_scalar( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_mul_scalar<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel mul scalar" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel mul scalar" );
};

template <typename type>
void cuda_kernel_div_scalar( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_div_scalar<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel div scalar" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel div scalar" );
};

template <typename type>
void cuda_kernel_div_scalar_inv( std::vector<int> & grid, std::vector<int> & block, 
                                 const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_div_scalar_inv<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel div scalar inv" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel div scalar inv" );
};

template <typename type>
void cuda_kernel_pow_scalar( std::vector<int> & grid, std::vector<int> & block, 
                             const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_pow_scalar<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel pow scalar" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel pow scalar" );
};

template <typename type>
void cuda_kernel_pow_scalar_inv( std::vector<int> & grid, std::vector<int> & block, 
                                 const type * vin, type * vout, type scalar, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_pow_scalar_inv<<<grd,blk>>>(vin, vout, scalar, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel pow scalar inv" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel pow scalar inv" );
};

template <typename type>
void cuda_kernel_add( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin1, const type * vin2, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_add<<<grd,blk>>>(vin1, vin2, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel add" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel add" );
};

template <typename type>
void cuda_kernel_sub( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin1, const type * vin2, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_sub<<<grd,blk>>>(vin1, vin2, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel sub" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel sub" );
};

template <typename type>
void cuda_kernel_mul( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin1, const type * vin2, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_mul<<<grd,blk>>>(vin1, vin2, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel mul" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel mul" );
};

template <typename type>
void cuda_kernel_div( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin1, const type * vin2, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_div<<<grd,blk>>>(vin1, vin2, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel div" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel div" );
};

template <typename type>
void cuda_kernel_pow( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin1, const type * vin2, type * vout, int n )
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_pow<<<grd,blk>>>(vin1, vin2, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel" );
};


// ===========================================
// Reduction Kernels
// ===========================================
template <typename type>
void cuda_kernel_sum( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin, type * vout, int n)
{
    // printf("kernel sum init\n");
    // printf("block: [%i, %i, %i]\n", block[0], block[1] , block[2]);
    // printf("grid: [%i, %i, %i]\n", grid[0], grid[1] , grid[2]);
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_sum<<<grd,blk>>>(vin, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel sum" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel sum" );
};

template <typename type>
void cuda_kernel_min( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin, type * vout, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_min<<<grd,blk>>>(vin, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel min" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel min" );
};

template <typename type>
void cuda_kernel_max( std::vector<int> & grid, std::vector<int> & block, 
                      const type * vin, type * vout, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_max<<<grd,blk>>>(vin, vout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel max" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel max" );
};

// ===========================================
// Image Kernels
// ===========================================
template <typename type>
void cuda_kernel_pad_2d( std::vector<int> & grid, std::vector<int> & block, 
                         const type * vin, type * vout, int start0, int start1,
                         int end0, int end1, int n0, int n1 )
{
    // printf("kernel pad init\n");
    // printf("block: [%i, %i, %i]\n", block[0], block[1] , block[2]);
    // printf("grid: [%i, %i, %i]\n", grid[0], grid[1] , grid[2]);
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_pad_2d<<<grd,blk>>>(vin, vout, start0, start1, end0, end1, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel pad 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel pad 2d" );
};

template <typename type>
void cuda_kernel_unpad_2d( std::vector<int> & grid, std::vector<int> & block, 
                           const type * vin, type * vout, int start0, int start1,
                           int end0, int end1, int n0, int n1 )
{
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_unpad_2d<<<grd,blk>>>(vin, vout, start0, start1, end0, end1, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel unpad 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel unpad 2d" );
};

template <typename type>
void cuda_kernel_pad_3d( std::vector<int> & grid, std::vector<int> & block, 
                         const type * vin, type * vout, int start0, int start1, int start2,
                         int end0, int end1, int end2, int n0, int n1, int n2 )
{
    // printf("kernel pad 3d init\n");
    // printf("block: [%i, %i, %i]\n", block[0], block[1] , block[2]);
    // printf("grid: [%i, %i, %i]\n", grid[0], grid[1] , grid[2]);
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_pad_3d<<<grd,blk>>>(vin, vout, start0, start1, start2, end0, end1, end2, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel pad 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel pad 3d" );
};

template <typename type>
void cuda_kernel_unpad_3d( std::vector<int> & grid, std::vector<int> & block, 
                           const type * vin, type * vout, int start0, int start1, int start2,
                         int end0, int end1, int end2, int n0, int n1, int n2 )
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_unpad_3d<<<grd,blk>>>(vin, vout, start0, start1, start2, end0, end1, end2, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel unpad 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel unpad 3d" );
};

template <typename type>
void cuda_kernel_grid_2d( std::vector<int> & grid, std::vector<int> & block, 
                          type * x, type * y, double * sod, 
                          int n0, int n1)
{
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_grid_2d<<<grd,blk>>>(x, y, sod, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel grid 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel grid 2d" );
};

template <typename type>
void cuda_kernel_grid_3d( std::vector<int> & grid, std::vector<int> & block,
                          type * x, type * y, type * z, double * sod, 
                          int n0, int n1, int n2)
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_grid_3d<<<grd,blk>>>(x, y, z, sod, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel grid 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel grid 3d" );
};

template <typename type>
void cuda_kernel_affine_2d( std::vector<int> & grid, std::vector<int> & block, 
                            const type * xin, const type * yin, 
                            type * xout, type * yout, 
                            const type * param, int n)
{
    // printf("kernel affine 2d init\n");
    // printf("block: [%i, %i, %i]\n", block[0], block[1] , block[2]);
    // printf("grid: [%i, %i, %i]\n", grid[0], grid[1] , grid[2]);
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_affine_2d<<<grd,blk>>>(xin, yin, xout, yout, param, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel affine 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel affine 2d" );
};

template <typename type>
void cuda_kernel_affine_3d( std::vector<int> & grid, std::vector<int> & block,
                            const type * xin, const type * yin, const type * zin,
                            type * xout, type * yout, type * zout,
                            const type * param, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_affine_3d<<<grd,blk>>>(xin, yin, zin, xout, yout, zout, param, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel affine 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel affine 3d" );
};

template <typename type>
void cuda_kernel_affine_sod_2d( std::vector<int> & grid, std::vector<int> & block,
                                const type * xin, const type * yin,
                                type * xout, type * yout,
                                const double * sod, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_affine_sod_2d<<<grd,blk>>>(xin, yin, xout, yout, sod, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel affine sod 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel affine sod 2d" );
};

template <typename type>
void cuda_kernel_affine_sod_3d( std::vector<int> & grid, std::vector<int> & block,
                                const type * xin, const type * yin, const type * zin,
                                type * xout, type * yout, type * zout,
                                const double * sod, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_affine_sod_3d<<<grd,blk>>>(xin, yin, zin, xout, yout, zout, sod, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel affine sod 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel affine sod 3d" );
};

template <typename type>
void cuda_kernel_dfield_2d( std::vector<int> & grid, std::vector<int> & block,
                            const type * xin, const type * yin,   // grid coordinates
                            const type * x, const type * y,       // vector field
                            type * xout, type * yout, int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_dfield_2d<<<grd,blk>>>(xin, yin, x, y, xout, yout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel dfield 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel dfield 2d" );
};

template <typename type>
void cuda_kernel_dfield_3d( std::vector<int> & grid, std::vector<int> & block,
                            const type * xin, const type * yin, const type * zin, // grid coordinates
                            const type * x, const type * y, const type * z,       // vector field
                            type * xout, type * yout, type * zout,                // output coordinates
                            int n)
{
    dim3 grd(grid[0]);
    dim3 blk(block[0]);
    kernel_dfield_3d<<<grd,blk>>>(xin, yin, zin, x, y, z, xout, yout, zout, n);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel dfield 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel dfield 3d" );
};

template <typename type>
void cuda_kernel_nearest_interpolation_2d( std::vector<int> & grid, std::vector<int> & block,
                                           const type * xo, const type * yo,
                                           const type * imgr, type * imgo,
                                           int w, int h,   //img ref width and height
                                           int n0, int n1) //img out dims
{
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_nearest_interpolation_2d<<<grd,blk>>>(xo, yo, imgr, imgo, w, h, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel nearest interpolation 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel nearest interpolation 2d" );
};

template <typename type>
void cuda_kernel_nearest_interpolation_3d( std::vector<int> & grid, std::vector<int> & block, 
                                           const type * xo, const type * yo, const type * zo, 
                                           const type * imgr, type * imgo,
                                           int w, int h, int l,    //img ref width, height and length
                                           int n0, int n1, int n2)
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_nearest_interpolation_3d<<<grd,blk>>>(xo, yo, zo, imgr, imgo, w, h, l, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel nearest interpolation 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel nearest interpolation 3d" );
};

template <typename type>
void cuda_kernel_linear_interpolation_2d( std::vector<int> & grid, std::vector<int> & block,
                                          const type * xo, const type * yo,
                                          const type * imgr, type * imgo,
                                          int w, int h,   //img ref width and height
                                          int n0, int n1) //img out dims
{
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_linear_interpolation_2d<<<grd,blk>>>(xo, yo, imgr, imgo, w, h, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel linear interpolation 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel linear interpolation 2d" );
};

template <typename type>
void cuda_kernel_linear_interpolation_3d( std::vector<int> & grid, std::vector<int> & block,
                                          const type * xo, const type * yo, const type * zo,
                                          const type * imgr, type * imgo,
                                          int w, int h, int l, //img ref width, height and length
                                          int n0, int n1, int n2)
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_linear_interpolation_3d<<<grd,blk>>>(xo, yo, zo, imgr, imgo, w, h, l, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel linear interpolation 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel linear interpolation 3d" );
};

template <typename type>
void cuda_kernel_gradientx( std::vector<int> & grid, std::vector<int> & block,
                            const type * imgr, type * imgo, 
                            int n0, int n1, int n2)
{
    dim3 grd;
    dim3 blk;
    if (block[2] == 0)
    {
        grd = dim3(grid[0],grid[1]);
        blk = dim3(block[0],block[1]);
    }
    else
    {
        grd = dim3(grid[0],grid[1],grid[2]);
        blk = dim3(block[0],block[1],block[2]);
    };

    kernel_gradientx<<<grd,blk>>>(imgr, imgo, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel gradient x" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel gradient x" );
};

template <typename type>
void cuda_kernel_gradienty( std::vector<int> & grid, std::vector<int> & block,
                            const type * imgr, type * imgo, 
                            int n0, int n1, int n2)
{
    dim3 grd;
    dim3 blk;
    if (block[2] == 0)
    {
        grd = dim3(grid[0],grid[1]);
        blk = dim3(block[0],block[1]);
    }
    else
    {
        grd = dim3(grid[0],grid[1],grid[2]);
        blk = dim3(block[0],block[1],block[2]);
    };
    kernel_gradienty<<<grd,blk>>>(imgr, imgo, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel gradient y" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel gradient y" );
};

template <typename type>
void cuda_kernel_gradientz( std::vector<int> & grid, std::vector<int> & block,
                            const type * imgr, type * imgo, 
                            int n0, int n1, int n2)
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_gradientz<<<grd,blk>>>(imgr, imgo, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel gradient z" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel gradient z" );
};

template <typename type>
void cuda_kernel_convolution_2d( std::vector<int> & grid, std::vector<int> & block,
                                 const type * imgr, const type * kern, //kernel width
                                 type * imgo, int kwidth, int n0, int n1)
{
    dim3 grd(grid[0],grid[1]);
    dim3 blk(block[0],block[1]);
    kernel_convolution_2d<<<grd,blk>>>(imgr, kern, imgo, kwidth, n0, n1);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel convolution 2d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel convolution 2d" );
};

template <typename type>
void cuda_kernel_convolution_3d( std::vector<int> & grid, std::vector<int> & block,
                                 const type * imgr, const type * kern, //kernel width
                                 type * imgo, int kwidth, int n0, int n1, int n2)
{
    dim3 grd(grid[0],grid[1],grid[2]);
    dim3 blk(block[0],block[1],block[2]);
    kernel_convolution_3d<<<grd,blk>>>(imgr, kern, imgo, kwidth, n0, n1, n2);
    imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel convolution 3d" );
    imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel convolution 3d" );
};


// template <typename type>
// void cuda_kernel_( std::vector<int> & grid, std::vector<int> & block, 
//                      )
// {
//     dim3 grd(grid[0],grid[1],grid[2]);
//     dim3 blk(block[0],block[1],block[2]);
//     kernel_<<<grd,blk>>>();
//     imart_assert_kernel( hipPeekAtLastError(), "Fail to run kernel" );
//     imart_assert_kernel( hipDeviceSynchronize(), "Fail to sync kernel" );
// };



// ===========================================
// Explicit instanciation
// ===========================================

// CASTINGS
template void cuda_kernel_cast<float,double>( std::vector<int> & grid, std::vector<int> & block, 
                       const float * vin, double * vout, int n );

template void cuda_kernel_cast<double,float>( std::vector<int> & grid, std::vector<int> & block, 
                       const double * vin, float * vout, int n );


template void cuda_kernel_cast<int,float>( std::vector<int> & grid, std::vector<int> & block, 
                       const int * vin, float * vout, int n );

template void cuda_kernel_cast<float,int>( std::vector<int> & grid, std::vector<int> & block, 
                       const float * vin, int * vout, int n );

template void cuda_kernel_cast<int,double>( std::vector<int> & grid, std::vector<int> & block, 
                       const int * vin, double * vout, int n );

template void cuda_kernel_cast<double,int>( std::vector<int> & grid, std::vector<int> & block, 
                       const double * vin, int * vout, int n );


template void cuda_kernel_cast<float,unsigned short>( std::vector<int> & grid, std::vector<int> & block, 
                       const float * vin, unsigned short * vout, int n );

template void cuda_kernel_cast<unsigned short,float>( std::vector<int> & grid, std::vector<int> & block, 
                       const unsigned short * vin, float * vout, int n );

template void cuda_kernel_cast<double,unsigned short>( std::vector<int> & grid, std::vector<int> & block, 
                       const double * vin, unsigned short * vout, int n );

template void cuda_kernel_cast<unsigned short,double>( std::vector<int> & grid, std::vector<int> & block, 
                       const unsigned short * vin, double * vout, int n );


template void cuda_kernel_cast<float,unsigned char>( std::vector<int> & grid, std::vector<int> & block, 
                       const float * vin, unsigned char * vout, int n );

template void cuda_kernel_cast<unsigned char,float>( std::vector<int> & grid, std::vector<int> & block, 
                       const unsigned char * vin, float * vout, int n );

template void cuda_kernel_cast<double,unsigned char>( std::vector<int> & grid, std::vector<int> & block, 
                       const double * vin, unsigned char * vout, int n );

template void cuda_kernel_cast<unsigned char,double>( std::vector<int> & grid, std::vector<int> & block, 
                       const unsigned char * vin, double * vout, int n );




template void cuda_kernel_assign<float>( std::vector<int> & grid, std::vector<int> & block,
                                         float * vin, float value, int n );

template void cuda_kernel_copy<float>( std::vector<int> & grid, std::vector<int> & block,
                                       const float * vin, float * vout, int n );

template void cuda_kernel_add<float>( std::vector<int> & grid, std::vector<int> & block,
                                      const float * vin1, const float * vin2, float * vout, int n );

template void cuda_kernel_sub<float>( std::vector<int> & grid, std::vector<int> & block,
                                      const float * vin1, const float * vin2, float * vout, int n );

template void cuda_kernel_mul<float>( std::vector<int> & grid, std::vector<int> & block,
                                      const float * vin1, const float * vin2, float * vout, int n );

template void cuda_kernel_div<float>( std::vector<int> & grid, std::vector<int> & block,
                                      const float * vin1, const float * vin2, float * vout, int n );

template void cuda_kernel_pow<float>( std::vector<int> & grid, std::vector<int> & block,
                                      const float * vin1, const float * vin2, float * vout, int n );

template void cuda_kernel_add_scalar<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_sub_scalar<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_sub_scalar_inv<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_mul_scalar<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_div_scalar<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_div_scalar_inv<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_pow_scalar<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_pow_scalar_inv<float>( std::vector<int> & grid, std::vector<int> & block, 
                                             const float * vin, float * vout, float scalar, int n );

template void cuda_kernel_sum<float>( std::vector<int> & grid, std::vector<int> & block, const float * vin, float * vout, int n );

template void cuda_kernel_min<float>( std::vector<int> & grid, std::vector<int> & block, const float * vin, float * vout, int n );

template void cuda_kernel_max<float>( std::vector<int> & grid, std::vector<int> & block, const float * vin, float * vout, int n );


template void cuda_kernel_pad_2d<float>( std::vector<int> & grid, std::vector<int> & block, 
                            const float * vin, float * vout, int start0, int start1, 
                            int end0, int end1, int n0, int n1 );

template void cuda_kernel_unpad_2d<float>( std::vector<int> & grid, std::vector<int> & block, 
                            const float * vin, float * vout, int start0, int start1,
                            int end0, int end1, int n0, int n1);

template void cuda_kernel_pad_3d<float>( std::vector<int> & grid, std::vector<int> & block, 
                         const float * vin, float * vout, int start0, int start1, int start2,
                         int end0, int end1, int end2, int n0, int n1, int n2);

template void cuda_kernel_unpad_3d<float>( std::vector<int> & grid, std::vector<int> & block, 
                            const float * vin, float * vout, int start0, int start1, int start2,
                            int end0, int end1, int end2, int n0, int n1, int n2);

template void cuda_kernel_grid_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                          float * x, float * y, double * sod, 
                          int n0, int n1);

template void cuda_kernel_grid_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                          float * x, float * y, float * z, double * sod, 
                          int n0, int n1, int n2);

template void cuda_kernel_affine_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * xin, const float * yin, 
                            float * xout, float * yout, 
                            const float * param, int n);

template void cuda_kernel_affine_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * xin, const float * yin, const float * zin,
                            float * xout, float * yout, float * zout,
                            const float * param, int n) ;

template void cuda_kernel_affine_sod_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                                const float * xin, const float * yin,
                                float * xout, float * yout,
                                const double * sod, int n);

template void cuda_kernel_affine_sod_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                                const float * xin, const float * yin, const float * zin,
                                float * xout, float * yout, float * zout,
                                const double * sod, int n);

template void cuda_kernel_dfield_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * xin, const float * yin,   // grid coordinates
                            const float * x, const float * y,       // vector field
                            float * xout, float * yout, int n);

template void cuda_kernel_dfield_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * xin, const float * yin, const float * zin, // grid coordinates
                            const float * x, const float * y, const float * z,       // vector field
                            float * xout, float * yout, float * zout, int n);

template void cuda_kernel_nearest_interpolation_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                                           const float * xo, const float * yo,
                                           const float * imgr, float * imgo,
                                           int w, int h,   //img ref width and height
                                           int n0, int n1); //img out dims

template void cuda_kernel_nearest_interpolation_3d<float>( std::vector<int> & grid, std::vector<int> & block, 
                                           const float * xo, const float * yo, const float * zo, 
                                           const float * imgr, float * imgo,
                                           int w, int h, int l,    //img ref width, height and length
                                           int n0, int n1, int n2 );

template void cuda_kernel_linear_interpolation_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                                          const float * xo, const float * yo,
                                          const float * imgr, float * imgo,
                                          int w, int h,   //img ref width and height
                                          int n0, int n1); //img out dims

template void cuda_kernel_linear_interpolation_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                                          const float * xo, const float * yo, const float * zo,
                                          const float * imgr, float * imgo,
                                          int w, int h, int l, //img ref width, height and length
                                          int n0, int n1, int n2);

template void cuda_kernel_gradientx<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * imgr, float * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradienty<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * imgr, float * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradientz<float>( std::vector<int> & grid, std::vector<int> & block,
                            const float * imgr, float * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_convolution_2d<float>( std::vector<int> & grid, std::vector<int> & block,
                                 const float * imgr, const float * kern, //kernel width
                                 float * imgo, int kwidth, int n0, int n1);

template void cuda_kernel_convolution_3d<float>( std::vector<int> & grid, std::vector<int> & block,
                                 const float * imgr, const float * kern, //kernel width
                                 float * imgo, int kwidth, int n0, int n1, int n2);






template void cuda_kernel_assign<double>( std::vector<int> & grid, std::vector<int> & block,
                                         double * vin, double value, int n );

template void cuda_kernel_copy<double>( std::vector<int> & grid, std::vector<int> & block,
                                       const double * vin, double * vout, int n );

template void cuda_kernel_add<double>( std::vector<int> & grid, std::vector<int> & block,
                                      const double * vin1, const double * vin2, double * vout, int n );

template void cuda_kernel_sub<double>( std::vector<int> & grid, std::vector<int> & block,
                                      const double * vin1, const double * vin2, double * vout, int n );

template void cuda_kernel_mul<double>( std::vector<int> & grid, std::vector<int> & block,
                                      const double * vin1, const double * vin2, double * vout, int n );

template void cuda_kernel_div<double>( std::vector<int> & grid, std::vector<int> & block,
                                      const double * vin1, const double * vin2, double * vout, int n );

template void cuda_kernel_pow<double>( std::vector<int> & grid, std::vector<int> & block,
                                      const double * vin1, const double * vin2, double * vout, int n );

template void cuda_kernel_add_scalar<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_sub_scalar<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_sub_scalar_inv<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_mul_scalar<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_div_scalar<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_div_scalar_inv<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_pow_scalar<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_pow_scalar_inv<double>( std::vector<int> & grid, std::vector<int> & block, 
                                             const double * vin, double * vout, double scalar, int n );

template void cuda_kernel_sum<double>( std::vector<int> & grid, std::vector<int> & block, const double * vin, double * vout, int n );

template void cuda_kernel_min<double>( std::vector<int> & grid, std::vector<int> & block, const double * vin, double * vout, int n );

template void cuda_kernel_max<double>( std::vector<int> & grid, std::vector<int> & block, const double * vin, double * vout, int n );

template void cuda_kernel_pad_2d<double>( std::vector<int> & grid, std::vector<int> & block, 
                            const double * vin, double * vout, int start0, int start1, 
                            int end0, int end1, int n0, int n1 );

template void cuda_kernel_unpad_2d<double>( std::vector<int> & grid, std::vector<int> & block, 
                            const double * vin, double * vout, int start0, int start1,
                            int end0, int end1, int n0, int n1);

template void cuda_kernel_pad_3d<double>( std::vector<int> & grid, std::vector<int> & block, 
                         const double * vin, double * vout, int start0, int start1, int start2,
                         int end0, int end1, int end2, int n0, int n1, int n2);

template void cuda_kernel_unpad_3d<double>( std::vector<int> & grid, std::vector<int> & block, 
                            const double * vin, double * vout, int start0, int start1, int start2,
                            int end0, int end1, int end2, int n0, int n1, int n2);

template void cuda_kernel_grid_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                          double * x, double * y, double * sod, 
                          int n0, int n1);

template void cuda_kernel_grid_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                          double * x, double * y, double * z, double * sod, 
                          int n0, int n1, int n2);

template void cuda_kernel_affine_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * xin, const double * yin, 
                            double * xout, double * yout, 
                            const double * param, int n);

template void cuda_kernel_affine_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * xin, const double * yin, const double * zin,
                            double * xout, double * yout, double * zout,
                            const double * param, int n) ;

template void cuda_kernel_affine_sod_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                                const double * xin, const double * yin,
                                double * xout, double * yout,
                                const double * sod, int n);

template void cuda_kernel_affine_sod_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                                const double * xin, const double * yin, const double * zin,
                                double * xout, double * yout, double * zout,
                                const double * sod, int n);

template void cuda_kernel_dfield_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * xin, const double * yin,   // grid coordinates
                            const double * x, const double * y,       // vector field
                            double * xout, double * yout, int n);

template void cuda_kernel_dfield_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * xin, const double * yin, const double * zin, // grid coordinates
                            const double * x, const double * y, const double * z,       // vector field
                            double * xout, double * yout, double * zout, int n);

template void cuda_kernel_nearest_interpolation_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                                           const double * xo, const double * yo,
                                           const double * imgr, double * imgo,
                                           int w, int h,   //img ref width and height
                                           int n0, int n1); //img out dims

template void cuda_kernel_nearest_interpolation_3d<double>( std::vector<int> & grid, std::vector<int> & block, 
                                           const double * xo, const double * yo, const double * zo, 
                                           const double * imgr, double * imgo,
                                           int w, int h, int l,    //img ref width, height and length
                                           int n0, int n1, int n2);

template void cuda_kernel_linear_interpolation_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                                          const double * xo, const double * yo,
                                          const double * imgr, double * imgo,
                                          int w, int h,   //img ref width and height
                                          int n0, int n1); //img out dims

template void cuda_kernel_linear_interpolation_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                                          const double * xo, const double * yo, const double * zo,
                                          const double * imgr, double * imgo,
                                          int w, int h, int l, //img ref width, height and length
                                          int n0, int n1, int n2);

template void cuda_kernel_gradientx<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * imgr, double * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradienty<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * imgr, double * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradientz<double>( std::vector<int> & grid, std::vector<int> & block,
                            const double * imgr, double * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_convolution_2d<double>( std::vector<int> & grid, std::vector<int> & block,
                                 const double * imgr, const double * kern, //kernel width
                                 double * imgo, int kwidth, int n0, int n1);

template void cuda_kernel_convolution_3d<double>( std::vector<int> & grid, std::vector<int> & block,
                                 const double * imgr, const double * kern, //kernel width
                                 double * imgo, int kwidth, int n0, int n1, int n2);





template void cuda_kernel_assign<int>( std::vector<int> & grid, std::vector<int> & block,
                                         int * vin, int value, int n );

template void cuda_kernel_copy<int>( std::vector<int> & grid, std::vector<int> & block,
                                       const int * vin, int * vout, int n );

template void cuda_kernel_add<int>( std::vector<int> & grid, std::vector<int> & block,
                                      const int * vin1, const int * vin2, int * vout, int n );

template void cuda_kernel_sub<int>( std::vector<int> & grid, std::vector<int> & block,
                                      const int * vin1, const int * vin2, int * vout, int n );

template void cuda_kernel_mul<int>( std::vector<int> & grid, std::vector<int> & block,
                                      const int * vin1, const int * vin2, int * vout, int n );

template void cuda_kernel_div<int>( std::vector<int> & grid, std::vector<int> & block,
                                      const int * vin1, const int * vin2, int * vout, int n );

template void cuda_kernel_pow<int>( std::vector<int> & grid, std::vector<int> & block,
                                      const int * vin1, const int * vin2, int * vout, int n );

template void cuda_kernel_add_scalar<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_sub_scalar<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_sub_scalar_inv<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_mul_scalar<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_div_scalar<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_div_scalar_inv<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_pow_scalar<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_pow_scalar_inv<int>( std::vector<int> & grid, std::vector<int> & block, 
                                             const int * vin, int * vout, int scalar, int n );

template void cuda_kernel_sum<int>( std::vector<int> & grid, std::vector<int> & block, const int * vin, int * vout, int n );

template void cuda_kernel_min<int>( std::vector<int> & grid, std::vector<int> & block, const int * vin, int * vout, int n );

template void cuda_kernel_max<int>( std::vector<int> & grid, std::vector<int> & block, const int * vin, int * vout, int n );


template void cuda_kernel_pad_2d<int>( std::vector<int> & grid, std::vector<int> & block, 
                            const int * vin, int * vout, int start0, int start1, 
                            int end0, int end1, int n0, int n1 );

template void cuda_kernel_unpad_2d<int>( std::vector<int> & grid, std::vector<int> & block, 
                            const int * vin, int * vout, int start0, int start1,
                            int end0, int end1, int n0, int n1 );

template void cuda_kernel_pad_3d<int>( std::vector<int> & grid, std::vector<int> & block, 
                         const int * vin, int * vout, int start0, int start1, int start2,
                         int end0, int end1, int end2, int n0, int n1, int n2 );

template void cuda_kernel_unpad_3d<int>( std::vector<int> & grid, std::vector<int> & block, 
                            const int * vin, int * vout, int start0, int start1, int start2,
                            int end0, int end1, int end2, int n0, int n1, int n2 );

template void cuda_kernel_grid_2d<int>( std::vector<int> & grid, std::vector<int> & block,
                          int * x, int * y, double * sod, 
                          int n0, int n1 );

template void cuda_kernel_grid_3d<int>( std::vector<int> & grid, std::vector<int> & block,
                          int * x, int * y, int * z, double * sod, 
                          int n0, int n1, int n2 );

template void cuda_kernel_affine_2d<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * xin, const int * yin, 
                            int * xout, int * yout, 
                            const int * param, int n );

template void cuda_kernel_affine_3d<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * xin, const int * yin, const int * zin,
                            int * xout, int * yout, int * zout,
                            const int * param, int n );

template void cuda_kernel_affine_sod_2d<int>( std::vector<int> & grid, std::vector<int> & block,
                                const int * xin, const int * yin,
                                int * xout, int * yout,
                                const double * sod, int n);

template void cuda_kernel_affine_sod_3d<int>( std::vector<int> & grid, std::vector<int> & block,
                                const int * xin, const int * yin, const int * zin,
                                int * xout, int * yout, int * zout,
                                const double * sod, int n );

template void cuda_kernel_dfield_2d<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * xin, const int * yin,   // grid coordinates
                            const int * x, const int * y,       // vector field
                            int * xout, int * yout, int n );

template void cuda_kernel_dfield_3d<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * xin, const int * yin, const int * zin, // grid coordinates
                            const int * x, const int * y, const int * z,       // vector field
                            int * xout, int * yout, int * zout, int n );

// template void cuda_kernel_nearest_interpolation_2d<int>( std::vector<int> & grid, std::vector<int> & block,
//                                            const int * xo, const int * yo,
//                                            const int * imgr, int * imgo,
//                                            int w, int h,   //img ref width and height
//                                            int n0, int n1); //img out dims

// template void cuda_kernel_nearest_interpolation_3d<int>( std::vector<int> & grid, std::vector<int> & block, 
//                                            const int * xo, const int * yo, const int * zo, 
//                                            const int * imgr, int * imgo,
//                                            int w, int h, int l,    //img ref width, height and length
//                                            int n0, int n1, int n2);

// template void cuda_kernel_linear_interpolation_2d<int>( std::vector<int> & grid, std::vector<int> & block,
//                                           const int * xo, const int * yo,
//                                           const int * imgr, int * imgo,
//                                           int w, int h,   //img ref width and height
//                                           int n0, int n1); //img out dims

// template void cuda_kernel_linear_interpolation_3d<int>( std::vector<int> & grid, std::vector<int> & block,
//                                           const int * xo, const int * yo, const int * zo,
//                                           const int * imgr, int * imgo,
//                                           int w, int h, int l, //img ref width, height and length
//                                           int n0, int n1, int n2);

template void cuda_kernel_gradientx<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * imgr, int * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradienty<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * imgr, int * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_gradientz<int>( std::vector<int> & grid, std::vector<int> & block,
                            const int * imgr, int * imgo, 
                            int n0, int n1, int n2);

template void cuda_kernel_convolution_2d<int>( std::vector<int> & grid, std::vector<int> & block,
                                 const int * imgr, const int * kern, //kernel width
                                 int * imgo, int kwidth, int n0, int n1);

template void cuda_kernel_convolution_3d<int>( std::vector<int> & grid, std::vector<int> & block,
                                 const int * imgr, const int * kern, //kernel width
                                 int * imgo, int kwidth, int n0, int n1, int n2);





template void cuda_kernel_assign<unsigned short>( std::vector<int> & grid, std::vector<int> & block,
                                         unsigned short * vin, unsigned short value, int n );

template void cuda_kernel_copy<unsigned short>( std::vector<int> & grid, std::vector<int> & block,
                                       const unsigned short * vin, unsigned short * vout, int n );


template void cuda_kernel_assign<unsigned int>( std::vector<int> & grid, std::vector<int> & block,
                                         unsigned int * vin, unsigned int value, int n );

template void cuda_kernel_copy<unsigned int>( std::vector<int> & grid, std::vector<int> & block,
                                       const unsigned int * vin, unsigned int * vout, int n );


template void cuda_kernel_assign<unsigned char>( std::vector<int> & grid, std::vector<int> & block,
                                         unsigned char * vin, unsigned char value, int n );

template void cuda_kernel_copy<unsigned char>( std::vector<int> & grid, std::vector<int> & block,
                                       const unsigned char * vin, unsigned char * vout, int n );


template void cuda_kernel_assign<short>( std::vector<int> & grid, std::vector<int> & block,
                                         short * vin, short value, int n );

template void cuda_kernel_copy<short>( std::vector<int> & grid, std::vector<int> & block,
                                       const short * vin, short * vout, int n );


template void cuda_kernel_assign<char>( std::vector<int> & grid, std::vector<int> & block,
                                         char * vin, char value, int n );

template void cuda_kernel_copy<char>( std::vector<int> & grid, std::vector<int> & block,
                                       const char * vin, char * vout, int n );
