#include "hip/hip_runtime.h"
/*
* @Author: jose
* @Date:   2020-08-24 00:00:00
* @Last Modified by:   jose
* @Last Modified time: 2020-08-24 00:00:00
*/

// local libs
#include "interface.cuh"

// ===========================================
// Check Errors
// ===========================================
#define imart_assert_cuda(status, msg) \
    imart_assert_cuda_error((status), __FILE__, __LINE__, msg);

void imart_assert_cuda_error(hipError_t code, const char *file, int line, const char* msg, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\n******* CUDA Error *******"\
                    "\n[Error] Information:\t%s"\
                    "\n[Error] Error code:\t%i"\
                    "\n[Error] Description:\t%s"\
                    "\n[Error] File:\t\t%s"\
                    "\n[Error] Line:\t\t%d\n",
                    msg, code, hipGetErrorString(code), file, line);
      if (abort) exit(code); 
   };
};

// ===========================================
// Kernels
// ===========================================
// __global__ void kernel_print(const char * msg)
__global__ void kernel_print()
{
    printf("[GPU] Hello!\n");
    // printf("[GPU] %s\n", msg);
};

// ===========================================
// Functions
// ===========================================
void cuda_check_gpu()
{
    int devicesCount;
    hipGetDeviceCount(&devicesCount);
    for(int deviceIndex = 0; deviceIndex < devicesCount; ++deviceIndex)
    {
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, deviceIndex);
        std::cout << "CUDA Device:\t" << deviceProperties.name << std::endl;
    }
};

void cuda_print()
{
    kernel_print<<<1, 1>>>();
    imart_assert_cuda( hipPeekAtLastError(), "Fail to run kernel print" );
    imart_assert_cuda( hipDeviceSynchronize(), "Fail to sync kernel print");
};

template <typename type>
void cuda_create_memory(type * & x, int size)
{
    imart_assert_cuda ( hipMalloc(&x, size*sizeof(type)), "Memory allocation" ); 
};

template <typename type>
void cuda_clean_memory(type * & x)
{
    imart_assert_cuda( hipFree(x), "Memory free" ); 
};

template <typename type>
void cuda_push_memory(type * x, type * data, int size, int offset)
{
    // printf("vector in:\n");
    // for(int i = 0; i < size; i++)
    //     printf("%f ",data[i]);
    imart_assert_cuda( hipMemcpy(x, data, size*sizeof(type), hipMemcpyHostToDevice), "Memory copy host to device" );
    // hipMemcpy(x + offset, data, size*sizeof(type), hipMemcpyHostToDevice);
};

template <typename type>
void cuda_push_memory(type * x, const type * data, int size, int offset)
{
    imart_assert_cuda( hipMemcpy(x, data, size*sizeof(type), hipMemcpyHostToDevice), "Memory copy host to device" );
};

template <typename type>
void cuda_pull_memory(type * x, type * data, int size, int offset)
{   
    // printf("pull\n");
    imart_assert_cuda( hipMemcpy(data, x, size*sizeof(type), hipMemcpyDeviceToHost), "Memory copy device to host" );
    // hipMemcpy(data, x + offset, size*sizeof(type), hipMemcpyDeviceToHost);
    // printf("vector out:\n");
    // for(int i = 0; i < size; i++)
    //     printf("%f ",data[i]);
};

// ===========================================
// Explicit instanciation
// ===========================================
template void cuda_create_memory<float>(float * & x, int size);
template void cuda_clean_memory<float>(float * & x);
template void cuda_push_memory<float>(float * x, float * data, int size, int offset);
template void cuda_push_memory<float>(float * x, const float * data, int size, int offset);
template void cuda_pull_memory<float>(float * x, float * data, int size, int offset);

template void cuda_create_memory<double>(double * & x, int size);
template void cuda_clean_memory<double>(double * & x);
template void cuda_push_memory<double>(double * x, double * data, int size, int offset);
template void cuda_push_memory<double>(double * x, const double * data, int size, int offset);
template void cuda_pull_memory<double>(double * x, double * data, int size, int offset);

template void cuda_create_memory<unsigned int>(unsigned int * & x, int size);
template void cuda_clean_memory<unsigned int>(unsigned int * & x);
template void cuda_push_memory<unsigned int>(unsigned int * x, unsigned int * data, int size, int offset);
template void cuda_push_memory<unsigned int>(unsigned int * x, const unsigned int * data, int size, int offset);
template void cuda_pull_memory<unsigned int>(unsigned int * x, unsigned int * data, int size, int offset);

template void cuda_create_memory<int>(int * & x, int size);
template void cuda_clean_memory<int>(int * & x);
template void cuda_push_memory<int>(int * x, int * data, int size, int offset);
template void cuda_push_memory<int>(int * x, const int * data, int size, int offset);
template void cuda_pull_memory<int>(int * x, int * data, int size, int offset);

template void cuda_create_memory<unsigned short>(unsigned short * & x, int size);
template void cuda_clean_memory<unsigned short>(unsigned short * & x);
template void cuda_push_memory<unsigned short>(unsigned short * x, unsigned short * data, int size, int offset);
template void cuda_push_memory<unsigned short>(unsigned short * x, const unsigned short * data, int size, int offset);
template void cuda_pull_memory<unsigned short>(unsigned short * x, unsigned short * data, int size, int offset);

template void cuda_create_memory<short>(short * & x, int size);
template void cuda_clean_memory<short>(short * & x);
template void cuda_push_memory<short>(short * x, short * data, int size, int offset);
template void cuda_push_memory<short>(short * x, const short * data, int size, int offset);
template void cuda_pull_memory<short>(short * x, short * data, int size, int offset);

template void cuda_create_memory<unsigned char>(unsigned char * & x, int size);
template void cuda_clean_memory<unsigned char>(unsigned char * & x);
template void cuda_push_memory<unsigned char>(unsigned char * x, unsigned char * data, int size, int offset);
template void cuda_push_memory<unsigned char>(unsigned char * x, const unsigned char * data, int size, int offset);
template void cuda_pull_memory<unsigned char>(unsigned char * x, unsigned char * data, int size, int offset);

template void cuda_create_memory<char>(char * & x, int size);
template void cuda_clean_memory<char>(char * & x);
template void cuda_push_memory<char>(char * x, char * data, int size, int offset);
template void cuda_push_memory<char>(char * x, const char * data, int size, int offset);
template void cuda_pull_memory<char>(char * x, char * data, int size, int offset);

